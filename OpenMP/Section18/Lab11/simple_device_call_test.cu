#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"

__global__ void add( int *a, int *b, int *c ) {
    *c = *a + *b;
}

int main( void ) {
    int a,b,c;
    int *dev_a,*dev_b,*dev_c;
    int size = sizeof(int);
    hipMalloc( (void**)&dev_a, size);
    hipMalloc( (void**)&dev_b, size);
    hipMalloc( (void**)&dev_c, size);
    a=100;
    b=500;
    hipMemcpy( dev_a, &a,size, hipMemcpyHostToDevice);
    hipMemcpy( dev_b, &b,size, hipMemcpyHostToDevice);
    add<<<1,1>>>( dev_a,dev_b, dev_c );
    hipMemcpy( &c, dev_c, size, hipMemcpyDeviceToHost);
    printf( "%i + %i = %i \n",a,b,c );
     hipFree( dev_a ) ;
     hipFree( dev_b ) ;
     hipFree( dev_c ) ;

    return 0;
}


